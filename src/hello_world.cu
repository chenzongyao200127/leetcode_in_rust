// hello_world.cu
// CUDA: Hello,World

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_world(void)
{
    printf("GPU: Hello world!\\n");
}

int main(int argc, char **argv)
{
    printf("CPU: Hello world!\\n");
    hello_world<<<1, 10>>>();
    hipDeviceReset(); // 如果没有这一行，无法从GPU输出 "hello world"
    return 0;
}
